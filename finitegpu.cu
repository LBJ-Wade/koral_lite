#include "hip/hip_runtime.h"
extern "C" {

#include "ko.h"

}

#include "kogpu.h"

#define TB_SIZE 64
#define ixTEST 13
#define iyTEST 21
#define izTEST 8
#define iiTEST 22222
#define ivTEST 0

// TODO get_u_device and get_ub_device are unnecessary, replace with get_u and get_ub 

/*
// get data value from array u_arr of the quantity indexed iv
// at the cell center indexed ix,iy,iz
// copied from get_u macro in ko.h
// iX(ix), iY(iy), iZ(iz) are macros defined in ko.h, that return either the index or 0
// depending on the problem dimension
__device__ ldouble get_u_device(ldouble* u_arr,int iv,int ix,int iy,int iz)
{
  ldouble u_out;
  u_out = u_arr[iv + (iX(ix)+(NGCX))*NV + \
		     (iY(iy)+(NGCY))*(SX)*NV + \
		     (iZ(iz)+(NGCZ))*(SY)*(SX)*NV];
  return u_out;
}

// get data value from array ub_arr of quantity indexed iv
// on the left wall of cell indexed ix,iy,iz in dimension idim
// copied from get_ub macro in ko.h
__device__ ldouble get_ub_device(ldouble* ub_arr, int iv, int ix, int iy, int iz, int idim)
{
  ldouble ub_out;
  ub_out = (idim==0 ? ub_arr[iv + (iX(ix)+(NGCX))*NV + \
				  (iY(iy)+(NGCY))*(SX+1)*NV + \
				  (iZ(iz)+(NGCZ))*(SY)*(SX+1)*NV] : \
	   (idim==1 ? ub_arr[iv + (iX(ix)+(NGCX))*NV + \
			          (iY(iy)+(NGCY))*(SX)*NV + \
		                  (iZ(iz)+(NGCZ))*(SY+1)*(SX)*NV] : \
	   (idim==2 ? ub_arr[iv + (iX(ix)+(NGCX))*NV + \
			          (iY(iy)+(NGCY))*(SX)*NV + \
			          (iZ(iz)+(NGCZ))*(SY)*(SX)*NV] : 0.)));
  return ub_out;
}
*/

// TODO replace get_xb and get_size_x  everywhere
// get grid coordinate on the cell wall indexed ic in dimension idim
// copied from get_xb macro in ko.h
__device__ ldouble get_xb_device(ldouble* xb_arr, int ic, int idim)
{
  ldouble xb_out;
  xb_out = (idim==0 ? xb_arr[ic+NG] :		     \
           (idim==1 ? xb_arr[ic+NG + NX+2*NG + 1] :  \
	   (idim==2 ? xb_arr[ic+NG + NX+2*NG +1 + NY+2*NG +1 ] : 0.)));

  return xb_out;
}

__device__ ldouble get_gKr_device(ldouble* gKr_arr, int i,int j, int k,
				  int ix, int iy, int iz)
{
  ldouble gKr_out = gKr_arr[i*4*4+j*4+k + (iX(ix)+(NGCX))*64 + \
				          (iY(iy)+(NGCY))*(SX)*64 + \
			                  (iZMET(iz)+(NGCZMET))*(SY)*(SX)*64];
  return gKr_out;
}

// get size of cell indexed ic in dimension idim
// copied from get_size_x in finite.c
__device__ ldouble get_size_x_device(ldouble* xb_arr, int ic, int idim)
{
  ldouble dx;
  dx = get_xb_device(xb_arr, ic+1,idim) - get_xb_device(xb_arr, ic, idim);
  return dx;
}


// fill geometry
__device__ int fill_geometry_device(int ix,int iy,int iz,void* geom,ldouble* g_arr, ldouble* G_arr)
{

  /*
  struct geometry *ggg 
    = (struct geometry *) geom;

  ggg->par=-1;
  ggg->ifacedim = -1;
  pick_g(ix,iy,iz,ggg->gg);
  pick_G(ix,iy,iz,ggg->GG);
  ggg->alpha=sqrt(-1./ggg->GG[0][0]);
  ggg->ix=ix;  ggg->iy=iy;  ggg->iz=iz;
  ggg->xxvec[0]=0.;
  ggg->xxvec[1]=get_x(ix,0);
  ggg->xxvec[2]=get_x(iy,1);
  ggg->xxvec[3]=get_x(iz,2);
  ggg->xx=ggg->xxvec[1];
  ggg->yy=ggg->xxvec[2];
  ggg->zz=ggg->xxvec[3];
  ggg->gdet=ggg->gg[3][4];
  ggg->gttpert=ggg->GG[3][4];
  ggg->coords=MYCOORDS;
  */
    
  return 0;
  
}


// Metric source term
// TODO: deleted RADIATION and SHEARINGBOX parts
__device__ int f_metric_source_term_device(int ix, int iy, int iz, ldouble* ss,
			                   ldouble* p_arr,
			                   ldouble* g_arr, ldouble* G_arr, ldouble* gKr_arr)
{
  int i;

  struct geometry geom;
  //fill_geometry(ix,iy,iz,&geom);
  fill_geometry_device(ix,iy,iz,&geom,g_arr,G_arr);
    
  //f_metric_source_term_arb(&get_u(p_arr,0,ix,iy,iz), &geom, ss, l_arr); // --> replace with code here, no need for two functions
  //struct geometry *geom = (struct geometry *) ggg;
  
  ldouble (*gg)[5],(*GG)[5],gdetu;
  ldouble *pp = &get_u(p_arr,0,ix,iy,iz);
  
  gg=geom.gg;
  GG=geom.GG;

  #if (GDETIN==0) //no metric determinant inside derivatives
  gdetu=1.;
  #else
  gdetu=geom.gdet;
  #endif

  ldouble dlgdet[3];
  dlgdet[0]=gg[0][4]; //D[gdet,x1]/gdet
  dlgdet[1]=gg[1][4]; //D[gdet,x2]/gdet
  dlgdet[2]=gg[2][4]; //D[gdet,x3]/gdet
  
  ldouble T[4][4];
  int ii, jj;
  //calculating stress energy tensor components
  //calc_Tij(pp,&geom,T); // TODO
  for(ii=0;ii<4;ii++)
    for(jj=0;jj<4;jj++)
      T[ii][jj]=0.;
  
  indices_2221_device(T,T,gg);


  /*
  for(ii=0;ii<4;ii++)
    for(jj=0;jj<4;jj++)
      {
	if(isnan(T[ii][jj])) 
	  {
	    printf("%d %d %e\n",ii,jj,T[ii][jj]);
	    my_err("nan in metric_source_terms\n");
	  }
      }
  */
  
  //converting to 4-velocity
  ldouble vcon[4],ucon[4];
  vcon[1]=pp[2];
  vcon[2]=pp[3];
  vcon[3]=pp[4];
  
  conv_vels_device(vcon,ucon,VELPRIM,VEL4,gg,GG); //TODO
  //ucon[0]=1.; ucon[1]=0.; ucon[2]=0.; ucon[2]=0.; //TODO 
  
  int k,l,iv;
  for(iv=0;iv<NV;iv++)
    ss[iv]=0.;  // zero out all source terms initially

  //terms with Christoffels
  for(k=0;k<4;k++)
    for(l=0;l<4;l++)
      {
	ss[1]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,0,k,ix,iy,iz);
	ss[2]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,1,k,ix,iy,iz);
	ss[3]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,2,k,ix,iy,iz);
	ss[4]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,3,k,ix,iy,iz);
      }

  //terms with dloggdet  
#if (GDETIN==0)
  for(l=1;l<4;l++)
    {
      ss[0]+=-dlgdet[l-1]*pp[RHO]*ucon[l];
      ss[1]+=-dlgdet[l-1]*(T[l][0]+pp[RHO]*ucon[l]);
      ss[2]+=-dlgdet[l-1]*(T[l][1]);
      ss[3]+=-dlgdet[l-1]*(T[l][2]);
      ss[4]+=-dlgdet[l-1]*(T[l][3]);
      ss[5]+=-dlgdet[l-1]*pp[ENTR]*ucon[l];
    }   
#endif
  
  return 0;
}

__global__ void calc_update_gpu_kernel(ldouble dtin, int Nloop_0, 
                                       int* loop_0_ix, int* loop_0_iy, int* loop_0_iz,
				       ldouble* xb_arr,
				       ldouble* flbx_arr, ldouble* flby_arr, ldouble* flbz_arr,
				       ldouble* u_arr, ldouble* p_arr,
				       ldouble* g_arr, ldouble* G_arr, ldouble* gKr_arr)
{

  int ii;
  int ix,iy,iz,iv;
  ldouble dx,dy,dz;
  ldouble flxl,flxr,flyl,flyr,flzl,flzr;
  ldouble val,du;
  ldouble ms[NV];
  //ldouble gs[NV]; //NOTE gs[NV] is for artifical sources, rarely used

  // get index for this thread
  // Nloop_0 is number of cells to update;
  // usually Nloop_0=NX*NY*NZ, but sometimes there are weird bcs inside domain 
  ii = blockIdx.x * blockDim.x + threadIdx.x;
  if(ii >= Nloop_0) return;
    
  // get indices from 1D arrays
  ix=loop_0_ix[ii];
  iy=loop_0_iy[ii];
  iz=loop_0_iz[ii]; 

  if(ii==iiTEST){
    printf("D   : %d %d %d %d\n",ii, ix,iy,iz);
  }

  // Source term
  // check if cell is active
  // NOTE: is_cell_active always returns 1 -- a placeholder function put in long ago
  
  if(0) //if(is_cell_active(ix,iy,iz)==0)
  {
    // Source terms applied only for active cells	  
    for(iv=0;iv<NV;iv++) ms[iv]=0.; 
  }
  else
  {
     // Get metric source terms ms[iv]
     // and any other source terms gs[iv] 

     f_metric_source_term_device(ix,iy,iz,ms, p_arr, g_arr, G_arr, gKr_arr);  //TODO: somewhat complicated
     //f_general_source_term(ix,iy,iz,gs); //NOTE: *very* rarely used, ignore for now
     for(iv=0;iv<NV;iv++)
     {
       ms[iv] = 0; // TODO: placeholder metric term of 0
       //ms[iv]+=gs[iv];
     }
  }
    
  // Get the cell size in the three directions
  dx = get_size_x_device(xb_arr,ix,0); //dx=get_size_x(ix,0);
  dy = get_size_x_device(xb_arr,iy,1); //dy=get_size_x(iy,1);
  dz = get_size_x_device(xb_arr,iz,2); //dz=get_size_x(iz,2);

  // test sizes 
  if(ii==iiTEST)
  {
    printf("D size_x 0 %e \n", get_size_x_device(xb_arr,ixTEST,0));
    printf("D size_x 1 %e \n", get_size_x_device(xb_arr,iyTEST,1));
    printf("D size_x 2 %e \n", get_size_x_device(xb_arr,izTEST,2));
  }
  
  //update all conserved according to fluxes and source terms      
  for(iv=0;iv<NV;iv++)
  {	

    // Get the initial value of the conserved quantity
    //val = get_u_device(u_arr,iv,ix,iy,iz);
    val = get_u(u_arr,iv,ix,iy,iz);
    
    if(ix==ixTEST && iy==iyTEST && iz==izTEST && iv==ivTEST)
      printf("D u: %e\n", val);
    
    // Get the fluxes on the six faces.
    // flbx, flby, flbz are the fluxes at the LEFT walls of cell ix, iy, iz.
    // To get the RIGHT fluxes, we need flbx(ix+1,iy,iz), etc.
    //flxl=get_ub_device(flbx_arr,iv,ix,iy,iz,0);
    //flxr=get_ub_device(flbx_arr,iv,ix+1,iy,iz,0);
    //flyl=get_ub_device(flby_arr,iv,ix,iy,iz,1);
    //flyr=get_ub_device(flby_arr,iv,ix,iy+1,iz,1);
    //flzl=get_ub_device(flbz_arr,iv,ix,iy,iz,2);
    //flzr=get_ub_device(flbz_arr,iv,ix,iy,iz+1,2);
    flxl=get_ub(flbx_arr,iv,ix,iy,iz,0);
    flxr=get_ub(flbx_arr,iv,ix+1,iy,iz,0);
    flyl=get_ub(flby_arr,iv,ix,iy,iz,1);
    flyr=get_ub(flby_arr,iv,ix,iy+1,iz,1);
    flzl=get_ub(flbz_arr,iv,ix,iy,iz,2);
    flzr=get_ub(flbz_arr,iv,ix,iy,iz+1,2);

    
    if(ix==ixTEST && iy==iyTEST && iz==izTEST && iv==ivTEST)
      printf("D fluxes: %e %e %e %e %e %e\n", flxl,flxr,flyl,flyr,flzl,flzr);

    // Compute Delta U from the six fluxes
    du = -(flxr-flxl)*dtin/dx - (flyr-flyl)*dtin/dy - (flzr-flzl)*dtin/dz;

    // Compute the new conserved by adding Delta U and the source term
    val += (du + ms[iv]*dtin);

    // Save the new conserved to memory
    
//#ifdef SKIPHDEVOLUTION
//  if(iv>=NVMHD)
//#endif
//#ifdef RADIATION
//#ifdef SKIPRADEVOLUTION
//#ifdef EVOLVEPHOTONNUMBER
//  if(iv!=EE && iv!=FX && iv!=FY && iv!=FZ && iv!=NF)
//#else
//  if(iv!=EE && iv!=FX && iv!=FY && iv!=FZ)
//#endif
//#endif  
//#endif  
//#ifdef SKIPHDBUTENERGY
//  if(iv>=NVMHD || iv==UU)
//#endif
	
    u_arr[iv] = val;
    //set_u(u,iv,ix,iy,iz,val);	 

  }  
}

int calc_update_gpu(ldouble dtin)
{

  int *d_loop0_ix,*d_loop0_iy,*d_loop0_iz;
  int *h_loop0_ix,*h_loop0_iy,*h_loop0_iz;
  ldouble *d_xb_arr;
  ldouble *d_u_arr, *d_p_arr;
  ldouble *d_g_arr, *d_G_arr, *d_gKr_arr;
  ldouble *d_flbx_arr,*d_flby_arr,*d_flbz_arr;
  
  hipError_t err = hipSuccess;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Allocate device arrays 
  
  // printf("ERROR (error code %s)!\n", hipGetErrorString(err));

  err = hipMalloc(&d_loop0_ix, sizeof(int)*Nloop_0);
  err = hipMalloc(&d_loop0_iy, sizeof(int)*Nloop_0);
  err = hipMalloc(&d_loop0_iz, sizeof(int)*Nloop_0);

  // NOTE: size of xb,flbx,flby,flbz is copied from initial malloc in misc.c
  // these need to be long long if the grid is on one tile and large (~256^3)
  long long Nxb    = (NX+1+NY+1+NZ+1+6*NG);
  long long Nprim  = (SX)*(SY)*(SZ)*NV;
  long long NfluxX = (SX+1)*(SY)*(SZ)*NV;
  long long NfluxY = (SX)*(SY+1)*(SZ)*NV;
  long long NfluxZ = (SX)*(SY)*(SZ+1)*NV;
  long long Nmet   = (SX)*(SY)*(SZMET)*gSIZE;
  long long Nkris=(SX)*(SY)*(SZMET)*64;
  
  err = hipMalloc(&d_xb_arr,   sizeof(ldouble)*Nxb);
  err = hipMalloc(&d_p_arr,    sizeof(ldouble)*Nprim);
  err = hipMalloc(&d_u_arr,    sizeof(ldouble)*Nprim);
  err = hipMalloc(&d_flbx_arr, sizeof(ldouble)*NfluxX);
  err = hipMalloc(&d_flby_arr, sizeof(ldouble)*NfluxY);
  err = hipMalloc(&d_flbz_arr, sizeof(ldouble)*NfluxZ);
  err = hipMalloc(&d_g_arr,    sizeof(ldouble)*Nmet);
  err = hipMalloc(&d_G_arr,    sizeof(ldouble)*Nmet);
  err = hipMalloc(&d_gKr_arr,  sizeof(ldouble)*Nkris);
  
  // Copy data to device arrays
  
  // NOTE: when we add more functions to device, most of these should only be copied once
  // Make 1D arrays of ix,iy,iz indicies and copy to device
  h_loop0_ix = (int*)malloc(sizeof(int)*Nloop_0);
  h_loop0_iy = (int*)malloc(sizeof(int)*Nloop_0);
  h_loop0_iz = (int*)malloc(sizeof(int)*Nloop_0);

  for(int ii=0; ii<Nloop_0; ii++){
    h_loop0_ix[ii] = loop_0[ii][0];     
    h_loop0_iy[ii] = loop_0[ii][1];     
    h_loop0_iz[ii] = loop_0[ii][2];
    if (ii==iiTEST) printf("H   :  %d %d %d %d\n",ii,h_loop0_ix[ii],h_loop0_iy[ii],h_loop0_iz[ii]) ;
  }

  err =  hipMemcpy(d_loop0_ix, h_loop0_ix, sizeof(int)*Nloop_0, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_loop0_iy, h_loop0_iy, sizeof(int)*Nloop_0, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_loop0_iz, h_loop0_iz, sizeof(int)*Nloop_0, hipMemcpyHostToDevice);

  free(h_loop0_ix);
  free(h_loop0_iy);
  free(h_loop0_iz);

  // copy grid boundary data from xb (global array) to device
  printf("H size_x 0 %e \n", get_size_x(ixTEST,0));
  printf("H size_x 1 %e \n", get_size_x(iyTEST,1));
  printf("H size_x 2 %e \n", get_size_x(izTEST,2));
  err =  hipMemcpy(d_xb_arr, xb, sizeof(ldouble)*Nxb, hipMemcpyHostToDevice);

  // copy conserved quantities from u (global array) to device
  printf("H u: %e \n", get_u(u,ivTEST,ixTEST,iyTEST,izTEST));
  err = hipMemcpy(d_u_arr, u, sizeof(ldouble)*Nprim, hipMemcpyHostToDevice);
  err = hipMemcpy(d_p_arr, p, sizeof(ldouble)*Nprim, hipMemcpyHostToDevice);

  // copy metric and Christoffels
  err = hipMemcpy(d_g_arr, g, sizeof(ldouble)*Nmet, hipMemcpyHostToDevice);
  err = hipMemcpy(d_G_arr, G, sizeof(ldouble)*Nmet, hipMemcpyHostToDevice);
  err = hipMemcpy(d_gKr_arr, gKr, sizeof(ldouble)*Nkris, hipMemcpyHostToDevice);
  
  // copy fluxes data from flbx,flby,flbz (global arrays) to device
  printf("H fluxes: %e %e %e %e %e %e\n",
	 get_ub(flbx,ivTEST,ixTEST,iyTEST,izTEST,0),
	 get_ub(flbx,ivTEST,ixTEST+1,iyTEST,izTEST,0),
         get_ub(flby,ivTEST,ixTEST,iyTEST,izTEST,1),
	 get_ub(flby,ivTEST,ixTEST,iyTEST+1,izTEST,1),
	 get_ub(flbz,ivTEST,ixTEST,iyTEST,izTEST,2),
	 get_ub(flbz,ivTEST,ixTEST,iyTEST,izTEST+1,2));
  err =  hipMemcpy(d_flbx_arr, flbx, sizeof(ldouble)*NfluxX, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_flby_arr, flby, sizeof(ldouble)*NfluxY, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_flbz_arr, flbz, sizeof(ldouble)*NfluxZ, hipMemcpyHostToDevice);

  // Launch calc_update_gpu_kernel

  int threadblocks = (Nloop_0 / TB_SIZE) + ((Nloop_0 % TB_SIZE)? 1:0);
  printf("\nTest %d\n", threadblocks); fflush(stdout);

  hipEventRecord(start);
  calc_update_gpu_kernel<<<threadblocks, TB_SIZE>>>(dtin, Nloop_0, 
						    d_loop0_ix, d_loop0_iy, d_loop0_iz,
						    d_xb_arr,
						    d_flbx_arr, d_flby_arr, d_flbz_arr,
						    d_u_arr, d_p_arr,
				                    d_g_arr, d_G_arr, d_gKr_arr);
  hipEventRecord(stop);
  err = hipPeekAtLastError();
  hipDeviceSynchronize(); //TODO: do we need this, does hipMemcpy synchrotnize?
  
  // printf("ERROR-Kernel (error code %s)!\n", hipGetErrorString(err));

  hipEventSynchronize(stop);
  float tms = 0.;
  hipEventElapsedTime(&tms, start,stop);
  printf("gpu update time: %0.2f \n",tms);
  
  // TODO Copy updated u back from device to global array u?
  //ldouble *u_tmp;
  //err = hipMemcpy(&u_tmp, d_u_arr, sizeof(ldouble)*Nprim, hipMemcpyDeviceToHost);
  
  // Free Device Memory
  hipFree(d_loop0_ix);
  hipFree(d_loop0_iy);
  hipFree(d_loop0_iz);
  
  hipFree(d_xb_arr);
  hipFree(d_flbx_arr);
  hipFree(d_flby_arr);
  hipFree(d_flbz_arr);
  hipFree(d_u_arr);
  hipFree(d_p_arr);
  hipFree(d_g_arr);
  hipFree(d_G_arr);
  hipFree(d_gKr_arr);

  // set global timestep dt
  dt = dtin;

  return 0;
}
