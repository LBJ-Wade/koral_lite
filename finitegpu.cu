#include "hip/hip_runtime.h"
extern "C" {

#include "ko.h"

}

#include "kogpu.h"

#define TB_SIZE 64
#define ixTEST 13
#define iyTEST 21
#define izTEST 8
#define iiTEST 22222
#define ivTEST 0

// TODO get_u_device and get_ub_device are unnecessary, replace with get_u and get_ub 

/*
// get data value from array u_arr of the quantity indexed iv
// at the cell center indexed ix,iy,iz
// copied from get_u macro in ko.h
// iX(ix), iY(iy), iZ(iz) are macros defined in ko.h, that return either the index or 0
// depending on the problem dimension
__device__ ldouble get_u_device(ldouble* u_arr,int iv,int ix,int iy,int iz)
{
  ldouble u_out;
  u_out = u_arr[iv + (iX(ix)+(NGCX))*NV + \
		     (iY(iy)+(NGCY))*(SX)*NV + \
		     (iZ(iz)+(NGCZ))*(SY)*(SX)*NV];
  return u_out;
}

// get data value from array ub_arr of quantity indexed iv
// on the left wall of cell indexed ix,iy,iz in dimension idim
// copied from get_ub macro in ko.h
__device__ ldouble get_ub_device(ldouble* ub_arr, int iv, int ix, int iy, int iz, int idim)
{
  ldouble ub_out;
  ub_out = (idim==0 ? ub_arr[iv + (iX(ix)+(NGCX))*NV + \
				  (iY(iy)+(NGCY))*(SX+1)*NV + \
				  (iZ(iz)+(NGCZ))*(SY)*(SX+1)*NV] : \
	   (idim==1 ? ub_arr[iv + (iX(ix)+(NGCX))*NV + \
			          (iY(iy)+(NGCY))*(SX)*NV + \
		                  (iZ(iz)+(NGCZ))*(SY+1)*(SX)*NV] : \
	   (idim==2 ? ub_arr[iv + (iX(ix)+(NGCX))*NV + \
			          (iY(iy)+(NGCY))*(SX)*NV + \
			          (iZ(iz)+(NGCZ))*(SY)*(SX)*NV] : 0.)));
  return ub_out;
}
*/

// TODO replace get_xb and get_size_x  everywhere
// get grid coordinate on the cell wall indexed ic in dimension idim
// copied from get_xb macro in ko.h
__device__ ldouble get_xb_device(ldouble* xb_arr, int ic, int idim)
{
  ldouble xb_out;
  xb_out = (idim==0 ? xb_arr[ic+NG] :		     \
           (idim==1 ? xb_arr[ic+NG + NX+2*NG + 1] :  \
	   (idim==2 ? xb_arr[ic+NG + NX+2*NG +1 + NY+2*NG +1 ] : 0.)));

  return xb_out;
}

__device__ ldouble get_gKr_device(ldouble* gKr_arr, int i,int j, int k,
				  int ix, int iy, int iz)
{
  ldouble gKr_out = gKr_arr[i*4*4+j*4+k + (iX(ix)+(NGCX))*64 + \
				          (iY(iy)+(NGCY))*(SX)*64 + \
			                  (iZMET(iz)+(NGCZMET))*(SY)*(SX)*64];
  return gKr_out;
}

// get size of cell indexed ic in dimension idim
// copied from get_size_x in finite.c
__device__ ldouble get_size_x_device(ldouble* xb_arr, int ic, int idim)
{
  ldouble dx;
  dx = get_xb_device(xb_arr, ic+1,idim) - get_xb_device(xb_arr, ic, idim);
  return dx;
}


__device__ __host__ int indices_21_device(ldouble A1[4],ldouble A2[4],ldouble gg[][5])
{
  int i;
  ldouble At[4];

  for(i=0;i<4;i++)
  {
    At[i]=0.;
  }

  for(i=0;i<4;i++)
  {
    int j;
    for(j = 0; j < 4; j++)
    {
      At[i] += A1[j] * gg[i][j];
    }
  }

  for (i=0; i<4; i++)
  {
    A2[i] = At[i];
  }
  
  return 0;
}

__device__ __host__ int indices_2211_device(ldouble T1[][4],ldouble T2[][4],ldouble gg[][5])
{
  int i,j,k,l;
  ldouble Tt[4][4];

  for(i=0;i<4;i++)
    {
      for(j=0;j<4;j++)
	{
	  Tt[i][j]=0.;
	  for(k=0;k<4;k++)
	    {
	      for(l=0;l<4;l++)
		{
		  Tt[i][j]+=T1[k][l]*gg[i][k]*gg[j][l];
		}	  
	    }
	}
    }

   for(i=0;i<4;i++)
    {
      for(j=0;j<4;j++)
	{
	  T2[i][j]=Tt[i][j];
	}
    }

  return 0;
}


__device__ __host__ int indices_2221_device(ldouble T1[][4],ldouble T2[][4],ldouble gg[][5])
{
  int i;
  ldouble Tt[4][4];

  for(i=0;i<4;i++)
  {
    int j;
      for(j=0;j<4;j++)
    {
      Tt[i][j]=0.;
    }
  }

  for(i=0;i<4;i++)
  {
    int j;
    for(j=0;j<4;j++)
    {
      int k;
      for(k=0;k<4;k++)
      {
        Tt[i][j]+=T1[i][k]*gg[k][j];
      }
    }
  }

  for(i=0;i<4;i++)
  {
    int j;
    for(j=0;j<4;j++)
    {
      T2[i][j]=Tt[i][j];
    }
  }

  return 0;
}

//*********************************************************************
//Takes primitives and computes ucon, ucov in VEL4 frame
//**********************************************************************

__device__ __host__ int calc_ucon_ucov_from_prims_device(ldouble *pr, void *ggg, ldouble *ucon, ldouble *ucov)
{
  struct geometry *geom
  = (struct geometry *) ggg;

  ucon[0] = 0.;
  ucon[1] = pr[VX];
  ucon[2] = pr[VY];
  ucon[3] = pr[VZ];
  
#ifdef NONRELMHD //only three-velocity used;
  fill_utinucon_device(ucon,geom->gg,geom->GG); 
  indices_21_device(ucon,ucov,geom->gg);
  return 0;
#endif
  
  conv_vels_both_device(ucon,ucon,ucov,VELPRIM,VEL4,geom->gg,geom->GG);
  
  return 0;
}


//*********************************************************************
//computes ut and then calculates ucon
//**********************************************************************

__device__ __host__ int conv_vels_device(ldouble *u1,ldouble *u2,int which1,int which2,ldouble gg[][5],ldouble GG[][5])
{
  
#ifdef NONRELMHD //only three-velocity used;
  u2[1]=u1[1];u2[2]=u1[2];u2[3]=u1[3];
  fill_utinucon_device(u2,gg,GG); 
  return 0;
#endif

  conv_vels_core_device(u1,u2,which1,which2,gg,GG,0);  // 0 means u^t is not yet known
  
  return 0;
}


//**********************************************************************
//calculates ucon, assuming ut is known
//**********************************************************************

__device__ __host__ int conv_vels_ut_device(ldouble *u1,ldouble *u2,int which1,int which2,ldouble gg[][5],ldouble GG[][5])
{
  
#ifdef NONRELMHD //only three-velocity used;
  u2[1]=u1[1];u2[2]=u1[2];u2[3]=u1[3];
  fill_utinucon_device(u2,gg,GG); 
  return 0;
#endif
  
  conv_vels_core_device(u1,u2,which1,which2,gg,GG,1);  // 1 means u^t is known
  
  return 0;
}


//**********************************************************************
//calculates both ucon and ucov, assuming ut is unknown 
//**********************************************************************

__device__ __host__ int conv_vels_both_device (ldouble *u1,ldouble *u2con,ldouble *u2cov,int which1,int which2,ldouble gg[][5],ldouble GG[][5])
{
  
#ifdef NONRELMHD //only three-velocity used;
  u2con[1]=u1[1];u2con[2]=u1[2];u2con[3]=u1[3];
  fill_utinucon_device(u2con,gg,GG); 
  indices_21_device(u2con,u2cov,gg);
  return 0;
#endif

  if(which2!=VEL4)
  {
    printf("conv_vels_both only works with which2==VEL4: %d -> %d\n",which1,which2);
    return -1;
  }
  
  conv_vels_core_device(u1,u2con,which1,which2,gg,GG,0); //0 means u^t is not yet known
  indices_21_device(u2con,u2cov,gg);

  return 0;
}


//**********************************************************************
//converts contravariant velocities u1 to contravariant u2con and covariant u2cov (if which2==VEL4)
// July 7, 17, Ramesh: Large reorganization
// sub-calculations done in fill_utinucon, fill_utinvel3. This version has been tested with test_con_vel.c.
//**********************************************************************

__device__ __host__ int conv_vels_core_device(ldouble *u1,ldouble *u2conout,int which1,int which2,
					      ldouble gg[][5],ldouble GG[][5],int utknown)
{
  
  int i,j;
  ldouble u2con[4];
  int verbose=0;
  if(verbose)
  {
    printf("conv_vels: %d -> %d\n",which1,which2);
    //print_4vector(u1); //TODO
  }

  /*************** VEL3 -> VEL3 ***************/
  if(which1==VEL3 && which2==VEL3)
  {
    for(i=0;i<4;i++) u2con[i]=u1[i];
  }
  
  /*************** VEL4 -> VEL4 ***************/
  else if(which1==VEL4 && which2==VEL4)
  {
    if(utknown==0)  // u^t is not known
    {
      fill_utinucon_device(u1, gg, GG);
    }
    
    for(i=0;i<4;i++)
    {
      u2con[i]=u1[i];
    }
  }
  
  /*************** VELR -> VELR ***************/
  else if(which1==VELR && which2==VELR)
  {
    for(i=0;i<4;i++)
    {
      u2con[i]=u1[i];
    }
  }
  
  /*************** VEL4 -> VEL3 ***************/
  else if(which1==VEL4 && which2==VEL3)
  {
    if(utknown==0)  // u^t is not known
    {
      fill_utinucon_device(u1, gg, GG);
    }
    
    for(i=0;i<4;i++)
    {
      u2con[i]=u1[i]/u1[0];
    }
  }
  
  /*************** VEL3 -> VEL4 ***************/
  else if(which1==VEL3 && which2==VEL4)
  {
    fill_utinvel3_device(u1, gg, GG);
    u2con[0] = u1[0];
    
    if(u2con[0] < 1. || isnan(u2con[0]))
    {
      printf("ut.nan in conv_vels(%d,%d) VEL3->VEL4 - returning error\n",which1,which2); //getchar();
      return -1;  
    }
    
    u2con[1] = u1[1] * u2con[0];
    u2con[2] = u1[2] * u2con[0];
    u2con[3] = u1[3] * u2con[0];
  }
  
  /*************** VEL3 -> VELR ***************/
  else if(which1==VEL3 && which2==VELR)
  {
    fill_utinvel3_device(u1, gg, GG);
    u2con[0] = u1[0];
    
    if(u2con[0] < 1. || isnan(u2con[0]))
    {
      printf("ut.nan in conv_vels(%d,%d) VEL3->VELR - returning error\n",which1,which2); //getchar();
      return -1;
    }
    
    //to 4-velocity
    u2con[1] = u1[1] * u2con[0];
    u2con[2] = u1[2] * u2con[0];
    u2con[3] = u1[3] * u2con[0];
    
    //to relative velocity
    for(i = 1; i < 4; i++)
    {
      u2con[i] = u2con[i] - u2con[0] * GG[0][i] / GG[0][0];
    }
  }
  
  /*************** VEL4 -> VELR ***************/
  else if (which1==VEL4 && which2==VELR)
  {
    if(utknown==0)  // u^t is not known
    {
      fill_utinucon_device(u1, gg, GG);
    }
    u2con[0] = u1[0];
    
    for(i = 1; i < 4; i++)
      u2con[i] = u1[i] - u2con[0] * GG[0][i] / GG[0][0];
  }

  /*************** VELR -> VEL4 ***************/
  else if (which1==VELR && which2==VEL4)
  {
    ldouble alpgam = calc_alpgam_device(u1, gg, GG);
    
    u2con[0]=-alpgam*GG[0][0];
    if(u2con[0]<0)
      u2con[0] = fabs(u2con[0]);
          
    u2con[1]=u1[1]-alpgam*GG[0][1];
    u2con[2]=u1[2]-alpgam*GG[0][2];
    u2con[3]=u1[3]-alpgam*GG[0][3];
  }
  
  /*************** VELR -> VEL3 ***************/
  else if (which1==VELR && which2==VEL3)
  {
    ldouble alpgam = calc_alpgam_device(u1, gg, GG);

    u2con[0]=-alpgam*GG[0][0];
    if(u2con[0]<0)
      u2con[0] = fabs(u2con[0]);

    u2con[1]=u1[1]/u2con[0] + GG[0][1]/GG[0][0];
    u2con[2]=u1[2]/u2con[0] + GG[0][2]/GG[0][0];
    u2con[3]=u1[3]/u2con[0] + GG[0][3]/GG[0][0];

  }

  /*************** not supported  ***************/
  else
  {
    //my_err("velocity conversion not supported.\n");
    return -1;
  }

  for (i = 0; i < 4; i++)
  {
    u2conout[i] = u2con[i];
  }

  if(verbose)
  {
    //print_4vector(u2con);
    printf("conv_vels done %d %d\n",which1,which2);
  }
  
  return 0;
}


//**********************************************************************
// July 9, 17, Ramesh: This is Andrew's version of alpgam, which ensures a positive quantity
//**********************************************************************

__device__ __host__ ldouble calc_alpgam_device(ldouble *u1, ldouble gg[][5], ldouble GG[][5])
{
  int i, j;
  ldouble qsq=0.;

  for(i=1;i<4;i++)
  {
    for(j=1;j<4;j++)
    {
      qsq+=u1[i]*u1[j]*gg[i][j];
    }
  }
  
  ldouble gamma2=(1. + qsq);
  ldouble alpha2=(-1./GG[0][0]);
  ldouble alpgam2=alpha2*gamma2;
  if(alpgam2<0.) {
    //printf("alpgam2.lt.0 in VELR->VEL4\n");
    return 1.;
  }
  ldouble alpgam=sqrt(alpgam2);
  
  return alpgam;
}

//**********************************************************************
// July 7, 17, Ramesh: Calculates u^t from spatial components of three-velocity VEL3
// We solve: ut^2 * (a + 2*b + c) = -1
//   where a = g00, b = g0i*ui, c = gij*ui*uj
//   solution: ut = sqrt(-1/(a + 2*b + c))
//**********************************************************************

__device__ __host__ int fill_utinvel3_device(ldouble *u1,double gg[][5],ldouble GG[][5])
{
  int i, j;
  ldouble a, b, c;
  a = gg[0][0];
  b = c = 0.;
  
  for(i = 1; i < 4; i++)
  {
    b += u1[i] * gg[0][i];
    
    for(j=1;j<4;j++)
    {
      c += u1[i] * u1[j] * gg[i][j];
    }
  }
  
  u1[0]=sqrt(-1. / (a + 2. * b + c));
  
  return 0;
}


//**********************************************************************
// Calculates u^t from spatial components of four-velocity u^mu
// July 7, 17, Ramesh: streamlined the code to improve efficiency
// We solve quadratic: a*ut^2 + 2*b*ut + c = 0
//   where a = g00, b = g0i*ui, c = 1 + gij*ui*uj
//   solution: ut = (-b +/- sqrt(b^2-a*c))/a
//**********************************************************************
__device__ __host__ int fill_utinucon_device(ldouble *u1,double gg[][5],ldouble GG[][5])
{
  ldouble a, b, c, delta;
  int i, j;
  
  a = gg[0][0];
  b = 0.;
  c = 1.;
  
  for(i = 1; i < 4; i++)
  {
    b += u1[i] * gg[0][i];
    
    for(j = 1; j < 4; j++)
    {
      c += u1[i] * u1[j] * gg[i][j];
    }
  }
  
  delta = b * b - a * c;  // Note: b here is half the usual value
  if (delta < 0.)
  {
    printf("delta.lt.0 in fill_utinucon\n");
    //my_err("delta.lt.0 in fill_utinucon\n");
  }
  
  if (a < 0.)
  {
    u1[0] = (-b - sqrt(delta)) / a;
  }
  else //this is in ergoregion
  {
    //ANDREW THIS IS WRONG, should be minus sign everywhere
    //u1[0] = (-b + sqrt(delta)) / a;
    u1[0] = (-b - sqrt(delta)) / a;
  }
  
  return 0;
}


__device__ __host__ void calc_bcon_bcov_bsq_from_4vel_device(ldouble *pr, ldouble *ucon, ldouble *ucov, void* ggg,
		                        		     ldouble *bcon, ldouble *bcov, ldouble *bsq)
{

  int j;
  struct geometry *geom
  = (struct geometry *) ggg;

  // First calculate bcon0
  bcon[0] = pr[B1]*ucov[1] + pr[B2] * ucov[2] + pr[B3] * ucov[3] ;
  
  // Then spatial components of bcon
  
#ifdef NONRELMHD
  for(j = 1; j < 4; j++)
    bcon[j] = pr[B1-1+j]; //b^i=B^i

#else  // relativistic case
  
  ldouble u0inv = 1. / ucon[0];
  for(j=1;j<4;j++)
    bcon[j] = (pr[B1-1+j] + bcon[0] * ucon[j]) * u0inv ;
  
#endif //NONRELMHD
  
  // Convert to bcov and calculate bsq
  indices_21_device(bcon, bcov, geom->gg);
  *bsq = dotB(bcon, bcov); //NOTE: preprocessor macro, ok

  return ;
}

__device__ __host__ int calc_Tij_device(ldouble *pp, void* ggg, ldouble T[][4])
{
  struct geometry *geom
    = (struct geometry *) ggg;

  ldouble (*gg)[5],(*GG)[5];
  gg=geom->gg;
  GG=geom->GG;

  int iv,i,j;
  ldouble rho=pp[RHO];
  ldouble uu=pp[UU];
  ldouble utcon[4],ucon[4],ucov[4];  
  ldouble bcon[4],bcov[4],bsq=0.;
  
  //converts to 4-velocity
  for(iv=1;iv<4;iv++)
    utcon[iv]=pp[1+iv];
  utcon[0]=0.;
  conv_vels_both_device(utcon,ucon,ucov,VELPRIM,VEL4,gg,GG);

#ifdef NONRELMHD
  ucon[0]=1.;
  ucov[0]=-1.;
#endif

#ifdef MAGNFIELD
  calc_bcon_bcov_bsq_from_4vel_device(pp, ucon, ucov, geom, bcon, bcov, &bsq); 
#else
  bcon[0]=bcon[1]=bcon[2]=bcon[3]=0.;
  bsq=0.;
#endif
  
  ldouble gamma=GAMMA;
  #ifdef CONSISTENTGAMMA
  //gamma=pick_gammagas(geom->ix,geom->iy,geom->iz); //TODO
  #endif
  ldouble gammam1=gamma-1.;

  ldouble p=(gamma-1.)*uu; 
  ldouble w=rho+uu+p;
  ldouble eta=w+bsq;
  ldouble ptot=p+0.5*bsq;

#ifndef NONRELMHD  
  for(i=0;i<4;i++)
    for(j=0;j<4;j++)
      T[i][j]=eta*ucon[i]*ucon[j] + ptot*GG[i][j] - bcon[i]*bcon[j];
#else
  
  ldouble v2=dot3nr(ucon,ucov); //TODO
  for(i=1;i<4;i++)
    for(j=1;j<4;j++)
      T[i][j]=(rho)*ucon[i]*ucon[j] + ptot*GG[i][j] - bcon[i]*bcon[j];

  T[0][0]=uu + bsq/2. + rho*v2/2.;
  for(i=1;i<4;i++)
    T[0][i]=T[i][0]=(T[0][0] + ptot) *ucon[i]*ucon[0] + ptot*GG[i][0] - bcon[i]*bcon[0];

#endif  // ifndef NONRELMHD

  return 0;
}



// fill geometry
__device__ int fill_geometry_device(int ix,int iy,int iz,void* geom,ldouble* g_arr, ldouble* G_arr)
{

  /*
  struct geometry *ggg 
    = (struct geometry *) geom;

  ggg->par=-1;
  ggg->ifacedim = -1;
  pick_g(ix,iy,iz,ggg->gg);
  pick_G(ix,iy,iz,ggg->GG);
  ggg->alpha=sqrt(-1./ggg->GG[0][0]);
  ggg->ix=ix;  ggg->iy=iy;  ggg->iz=iz;
  ggg->xxvec[0]=0.;
  ggg->xxvec[1]=get_x(ix,0);
  ggg->xxvec[2]=get_x(iy,1);
  ggg->xxvec[3]=get_x(iz,2);
  ggg->xx=ggg->xxvec[1];
  ggg->yy=ggg->xxvec[2];
  ggg->zz=ggg->xxvec[3];
  ggg->gdet=ggg->gg[3][4];
  ggg->gttpert=ggg->GG[3][4];
  ggg->coords=MYCOORDS;
  */
    
  return 0;
  
}


// Metric source term
// TODO: deleted RADIATION and SHEARINGBOX parts
__device__ int f_metric_source_term_device(int ix, int iy, int iz, ldouble* ss,
			                   ldouble* p_arr,
			                   ldouble* g_arr, ldouble* G_arr, ldouble* gKr_arr)
{
  int i;

  struct geometry geom;
  //fill_geometry(ix,iy,iz,&geom);
  fill_geometry_device(ix,iy,iz,&geom,g_arr,G_arr);
    
  //f_metric_source_term_arb(&get_u(p_arr,0,ix,iy,iz), &geom, ss, l_arr); // --> replace with code here, no need for two functions
  //struct geometry *geom = (struct geometry *) ggg;
  
  ldouble (*gg)[5],(*GG)[5],gdetu;
  ldouble *pp = &get_u(p_arr,0,ix,iy,iz);
  
  gg=geom.gg;
  GG=geom.GG;

  #if (GDETIN==0) //no metric determinant inside derivatives
  gdetu=1.;
  #else
  gdetu=geom.gdet;
  #endif

  ldouble dlgdet[3];
  dlgdet[0]=gg[0][4]; //D[gdet,x1]/gdet
  dlgdet[1]=gg[1][4]; //D[gdet,x2]/gdet
  dlgdet[2]=gg[2][4]; //D[gdet,x3]/gdet
  
  ldouble T[4][4];
  int ii, jj;
  //calculating stress energy tensor components
  calc_Tij_device(pp,&geom,T); // TODO
  for(ii=0;ii<4;ii++)
    for(jj=0;jj<4;jj++)
      T[ii][jj]=0.;
  
  indices_2221_device(T,T,gg);


  /*
  for(ii=0;ii<4;ii++)
    for(jj=0;jj<4;jj++)
      {
	if(isnan(T[ii][jj])) 
	  {
	    printf("%d %d %e\n",ii,jj,T[ii][jj]);
	    my_err("nan in metric_source_terms\n");
	  }
      }
  */
  
  //converting to 4-velocity
  ldouble vcon[4],ucon[4];
  vcon[1]=pp[2];
  vcon[2]=pp[3];
  vcon[3]=pp[4];
  
  conv_vels_device(vcon,ucon,VELPRIM,VEL4,gg,GG); //TODO
  //ucon[0]=1.; ucon[1]=0.; ucon[2]=0.; ucon[2]=0.; //TODO 
  
  int k,l,iv;
  for(iv=0;iv<NV;iv++)
    ss[iv]=0.;  // zero out all source terms initially

  //terms with Christoffels
  for(k=0;k<4;k++)
    for(l=0;l<4;l++)
      {
	ss[1]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,0,k,ix,iy,iz);
	ss[2]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,1,k,ix,iy,iz);
	ss[3]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,2,k,ix,iy,iz);
	ss[4]+=gdetu*T[k][l]*get_gKr_device(gKr_arr,l,3,k,ix,iy,iz);
      }

  //terms with dloggdet  
#if (GDETIN==0)
  for(l=1;l<4;l++)
    {
      ss[0]+=-dlgdet[l-1]*pp[RHO]*ucon[l];
      ss[1]+=-dlgdet[l-1]*(T[l][0]+pp[RHO]*ucon[l]);
      ss[2]+=-dlgdet[l-1]*(T[l][1]);
      ss[3]+=-dlgdet[l-1]*(T[l][2]);
      ss[4]+=-dlgdet[l-1]*(T[l][3]);
      ss[5]+=-dlgdet[l-1]*pp[ENTR]*ucon[l];
    }   
#endif
  
  return 0;
}

__global__ void calc_update_gpu_kernel(ldouble dtin, int Nloop_0, 
                                       int* loop_0_ix, int* loop_0_iy, int* loop_0_iz,
				       ldouble* xb_arr,
				       ldouble* flbx_arr, ldouble* flby_arr, ldouble* flbz_arr,
				       ldouble* u_arr, ldouble* p_arr,
				       ldouble* g_arr, ldouble* G_arr, ldouble* gKr_arr)
{

  int ii;
  int ix,iy,iz,iv;
  ldouble dx,dy,dz;
  ldouble flxl,flxr,flyl,flyr,flzl,flzr;
  ldouble val,du;
  ldouble ms[NV];
  //ldouble gs[NV]; //NOTE gs[NV] is for artifical sources, rarely used

  // get index for this thread
  // Nloop_0 is number of cells to update;
  // usually Nloop_0=NX*NY*NZ, but sometimes there are weird bcs inside domain 
  ii = blockIdx.x * blockDim.x + threadIdx.x;
  if(ii >= Nloop_0) return;
    
  // get indices from 1D arrays
  ix=loop_0_ix[ii];
  iy=loop_0_iy[ii];
  iz=loop_0_iz[ii]; 

  if(ii==iiTEST){
    printf("D   : %d %d %d %d\n",ii, ix,iy,iz);
  }

  // Source term
  // check if cell is active
  // NOTE: is_cell_active always returns 1 -- a placeholder function put in long ago
  
  if(0) //if(is_cell_active(ix,iy,iz)==0)
  {
    // Source terms applied only for active cells	  
    for(iv=0;iv<NV;iv++) ms[iv]=0.; 
  }
  else
  {
     // Get metric source terms ms[iv]
     // and any other source terms gs[iv] 

     f_metric_source_term_device(ix,iy,iz,ms, p_arr, g_arr, G_arr, gKr_arr);  //TODO: somewhat complicated
     //f_general_source_term(ix,iy,iz,gs); //NOTE: *very* rarely used, ignore for now
     for(iv=0;iv<NV;iv++)
     {
       ms[iv] = 0; // TODO: placeholder metric term of 0
       //ms[iv]+=gs[iv];
     }
  }
    
  // Get the cell size in the three directions
  dx = get_size_x_device(xb_arr,ix,0); //dx=get_size_x(ix,0);
  dy = get_size_x_device(xb_arr,iy,1); //dy=get_size_x(iy,1);
  dz = get_size_x_device(xb_arr,iz,2); //dz=get_size_x(iz,2);

  // test sizes 
  if(ii==iiTEST)
  {
    printf("D size_x 0 %e \n", get_size_x_device(xb_arr,ixTEST,0));
    printf("D size_x 1 %e \n", get_size_x_device(xb_arr,iyTEST,1));
    printf("D size_x 2 %e \n", get_size_x_device(xb_arr,izTEST,2));
  }
  
  //update all conserved according to fluxes and source terms      
  for(iv=0;iv<NV;iv++)
  {	

    // Get the initial value of the conserved quantity
    //val = get_u_device(u_arr,iv,ix,iy,iz);
    val = get_u(u_arr,iv,ix,iy,iz);
    
    if(ix==ixTEST && iy==iyTEST && iz==izTEST && iv==ivTEST)
      printf("D u: %e\n", val);
    
    // Get the fluxes on the six faces.
    // flbx, flby, flbz are the fluxes at the LEFT walls of cell ix, iy, iz.
    // To get the RIGHT fluxes, we need flbx(ix+1,iy,iz), etc.
    //flxl=get_ub_device(flbx_arr,iv,ix,iy,iz,0);
    //flxr=get_ub_device(flbx_arr,iv,ix+1,iy,iz,0);
    //flyl=get_ub_device(flby_arr,iv,ix,iy,iz,1);
    //flyr=get_ub_device(flby_arr,iv,ix,iy+1,iz,1);
    //flzl=get_ub_device(flbz_arr,iv,ix,iy,iz,2);
    //flzr=get_ub_device(flbz_arr,iv,ix,iy,iz+1,2);
    flxl=get_ub(flbx_arr,iv,ix,iy,iz,0);
    flxr=get_ub(flbx_arr,iv,ix+1,iy,iz,0);
    flyl=get_ub(flby_arr,iv,ix,iy,iz,1);
    flyr=get_ub(flby_arr,iv,ix,iy+1,iz,1);
    flzl=get_ub(flbz_arr,iv,ix,iy,iz,2);
    flzr=get_ub(flbz_arr,iv,ix,iy,iz+1,2);

    
    if(ix==ixTEST && iy==iyTEST && iz==izTEST && iv==ivTEST)
      printf("D fluxes: %e %e %e %e %e %e\n", flxl,flxr,flyl,flyr,flzl,flzr);

    // Compute Delta U from the six fluxes
    du = -(flxr-flxl)*dtin/dx - (flyr-flyl)*dtin/dy - (flzr-flzl)*dtin/dz;

    // Compute the new conserved by adding Delta U and the source term
    val += (du + ms[iv]*dtin);

    // Save the new conserved to memory
    
//#ifdef SKIPHDEVOLUTION
//  if(iv>=NVMHD)
//#endif
//#ifdef RADIATION
//#ifdef SKIPRADEVOLUTION
//#ifdef EVOLVEPHOTONNUMBER
//  if(iv!=EE && iv!=FX && iv!=FY && iv!=FZ && iv!=NF)
//#else
//  if(iv!=EE && iv!=FX && iv!=FY && iv!=FZ)
//#endif
//#endif  
//#endif  
//#ifdef SKIPHDBUTENERGY
//  if(iv>=NVMHD || iv==UU)
//#endif
	
    u_arr[iv] = val;
    //set_u(u,iv,ix,iy,iz,val);	 

  }  
}

int calc_update_gpu(ldouble dtin)
{

  int *d_loop0_ix,*d_loop0_iy,*d_loop0_iz;
  int *h_loop0_ix,*h_loop0_iy,*h_loop0_iz;
  ldouble *d_xb_arr;
  ldouble *d_u_arr, *d_p_arr;
  ldouble *d_g_arr, *d_G_arr, *d_gKr_arr;
  ldouble *d_flbx_arr,*d_flby_arr,*d_flbz_arr;
  
  hipError_t err = hipSuccess;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Allocate device arrays 
  
  // printf("ERROR (error code %s)!\n", hipGetErrorString(err));

  err = hipMalloc(&d_loop0_ix, sizeof(int)*Nloop_0);
  err = hipMalloc(&d_loop0_iy, sizeof(int)*Nloop_0);
  err = hipMalloc(&d_loop0_iz, sizeof(int)*Nloop_0);

  // NOTE: size of xb,flbx,flby,flbz is copied from initial malloc in misc.c
  // these need to be long long if the grid is on one tile and large (~256^3)
  long long Nxb    = (NX+1+NY+1+NZ+1+6*NG);
  long long Nprim  = (SX)*(SY)*(SZ)*NV;
  long long NfluxX = (SX+1)*(SY)*(SZ)*NV;
  long long NfluxY = (SX)*(SY+1)*(SZ)*NV;
  long long NfluxZ = (SX)*(SY)*(SZ+1)*NV;
  long long Nmet   = (SX)*(SY)*(SZMET)*gSIZE;
  long long Nkris=(SX)*(SY)*(SZMET)*64;
  
  err = hipMalloc(&d_xb_arr,   sizeof(ldouble)*Nxb);
  err = hipMalloc(&d_p_arr,    sizeof(ldouble)*Nprim);
  err = hipMalloc(&d_u_arr,    sizeof(ldouble)*Nprim);
  err = hipMalloc(&d_flbx_arr, sizeof(ldouble)*NfluxX);
  err = hipMalloc(&d_flby_arr, sizeof(ldouble)*NfluxY);
  err = hipMalloc(&d_flbz_arr, sizeof(ldouble)*NfluxZ);
  err = hipMalloc(&d_g_arr,    sizeof(ldouble)*Nmet);
  err = hipMalloc(&d_G_arr,    sizeof(ldouble)*Nmet);
  err = hipMalloc(&d_gKr_arr,  sizeof(ldouble)*Nkris);
  
  // Copy data to device arrays
  
  // NOTE: when we add more functions to device, most of these should only be copied once
  // Make 1D arrays of ix,iy,iz indicies and copy to device
  h_loop0_ix = (int*)malloc(sizeof(int)*Nloop_0);
  h_loop0_iy = (int*)malloc(sizeof(int)*Nloop_0);
  h_loop0_iz = (int*)malloc(sizeof(int)*Nloop_0);

  for(int ii=0; ii<Nloop_0; ii++){
    h_loop0_ix[ii] = loop_0[ii][0];     
    h_loop0_iy[ii] = loop_0[ii][1];     
    h_loop0_iz[ii] = loop_0[ii][2];
    if (ii==iiTEST) printf("H   :  %d %d %d %d\n",ii,h_loop0_ix[ii],h_loop0_iy[ii],h_loop0_iz[ii]) ;
  }

  err =  hipMemcpy(d_loop0_ix, h_loop0_ix, sizeof(int)*Nloop_0, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_loop0_iy, h_loop0_iy, sizeof(int)*Nloop_0, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_loop0_iz, h_loop0_iz, sizeof(int)*Nloop_0, hipMemcpyHostToDevice);

  free(h_loop0_ix);
  free(h_loop0_iy);
  free(h_loop0_iz);

  // copy grid boundary data from xb (global array) to device
  printf("H size_x 0 %e \n", get_size_x(ixTEST,0));
  printf("H size_x 1 %e \n", get_size_x(iyTEST,1));
  printf("H size_x 2 %e \n", get_size_x(izTEST,2));
  err =  hipMemcpy(d_xb_arr, xb, sizeof(ldouble)*Nxb, hipMemcpyHostToDevice);

  // copy conserved quantities from u (global array) to device
  printf("H u: %e \n", get_u(u,ivTEST,ixTEST,iyTEST,izTEST));
  err = hipMemcpy(d_u_arr, u, sizeof(ldouble)*Nprim, hipMemcpyHostToDevice);
  err = hipMemcpy(d_p_arr, p, sizeof(ldouble)*Nprim, hipMemcpyHostToDevice);

  // copy metric and Christoffels
  err = hipMemcpy(d_g_arr, g, sizeof(ldouble)*Nmet, hipMemcpyHostToDevice);
  err = hipMemcpy(d_G_arr, G, sizeof(ldouble)*Nmet, hipMemcpyHostToDevice);
  err = hipMemcpy(d_gKr_arr, gKr, sizeof(ldouble)*Nkris, hipMemcpyHostToDevice);
  
  // copy fluxes data from flbx,flby,flbz (global arrays) to device
  printf("H fluxes: %e %e %e %e %e %e\n",
	 get_ub(flbx,ivTEST,ixTEST,iyTEST,izTEST,0),
	 get_ub(flbx,ivTEST,ixTEST+1,iyTEST,izTEST,0),
         get_ub(flby,ivTEST,ixTEST,iyTEST,izTEST,1),
	 get_ub(flby,ivTEST,ixTEST,iyTEST+1,izTEST,1),
	 get_ub(flbz,ivTEST,ixTEST,iyTEST,izTEST,2),
	 get_ub(flbz,ivTEST,ixTEST,iyTEST,izTEST+1,2));
  err =  hipMemcpy(d_flbx_arr, flbx, sizeof(ldouble)*NfluxX, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_flby_arr, flby, sizeof(ldouble)*NfluxY, hipMemcpyHostToDevice);
  err =  hipMemcpy(d_flbz_arr, flbz, sizeof(ldouble)*NfluxZ, hipMemcpyHostToDevice);

  // Launch calc_update_gpu_kernel

  int threadblocks = (Nloop_0 / TB_SIZE) + ((Nloop_0 % TB_SIZE)? 1:0);
  printf("\nTest %d\n", threadblocks); fflush(stdout);

  hipEventRecord(start);
  calc_update_gpu_kernel<<<threadblocks, TB_SIZE>>>(dtin, Nloop_0, 
						    d_loop0_ix, d_loop0_iy, d_loop0_iz,
						    d_xb_arr,
						    d_flbx_arr, d_flby_arr, d_flbz_arr,
						    d_u_arr, d_p_arr,
				                    d_g_arr, d_G_arr, d_gKr_arr);
  hipEventRecord(stop);
  err = hipPeekAtLastError();
  hipDeviceSynchronize(); //TODO: do we need this, does hipMemcpy synchrotnize?
  
  // printf("ERROR-Kernel (error code %s)!\n", hipGetErrorString(err));

  hipEventSynchronize(stop);
  float tms = 0.;
  hipEventElapsedTime(&tms, start,stop);
  printf("gpu update time: %0.2f \n",tms);
  
  // TODO Copy updated u back from device to global array u?
  //ldouble *u_tmp;
  //err = hipMemcpy(&u_tmp, d_u_arr, sizeof(ldouble)*Nprim, hipMemcpyDeviceToHost);
  
  // Free Device Memory
  hipFree(d_loop0_ix);
  hipFree(d_loop0_iy);
  hipFree(d_loop0_iz);
  
  hipFree(d_xb_arr);
  hipFree(d_flbx_arr);
  hipFree(d_flby_arr);
  hipFree(d_flbz_arr);
  hipFree(d_u_arr);
  hipFree(d_p_arr);
  hipFree(d_g_arr);
  hipFree(d_G_arr);
  hipFree(d_gKr_arr);

  // set global timestep dt
  dt = dtin;

  return 0;
}
